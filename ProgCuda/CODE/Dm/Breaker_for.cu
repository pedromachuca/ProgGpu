#include <stdlib.h>
#include <sys/time.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ int endLoop=0;

__device__ __host__ int get_keypair(char* tab, int length, int first_char, int last_char){

	int sum=0;

	int pow=1;
	int i=0;
	
	for(i=0; i<length; i++){
		sum+=tab[i]*pow;
		pow*=(last_char - first_char);	
	}
	return sum;
}


__host__ __device__ int check_keypairs(int crypt, int test)
{
	if(crypt == test){
		return 1;
	}
	else{
		return 0;
	}
}


__global__ void kernel( int *crypted, int length, int first_char, int last_char, double max_iter){

	int loop_size = last_char - first_char;
	
	int i =blockIdx.x * blockDim.x + threadIdx.x;
	int total = (blockDim.x * gridDim.x);	

	char *tab = (char*)malloc(sizeof(char)*(length+1));
	tab[length]='\0';
	int j;
	for(j=0; j<length; j++) tab[j] = first_char;

	int current_keypair;
	int pow=0;	

	for(int j=i; j<max_iter; j+=total){
		pow=1;
		for(int x=0; x<length; x++){
			tab[x] = ((j/pow) % loop_size) + first_char;
			pow*=loop_size;
		}
		current_keypair = get_keypair(tab, length, first_char, last_char);
		
		if( check_keypairs(*crypted, current_keypair) ) {
			printf( "password found: %s\n", tab );
			endLoop=1;
		}
		if(endLoop==1){
			j=max_iter;
		}	
	}	
}


int main( int argc, char** argv ) {
	char* password; 
	int first_char, last_char;
	float t1, t2; 
	
	//unsigned long cmp;
	
	if( argc == 1 ) {
		password = "A$4c";
		first_char = 32;
		last_char = 126;
		/* ---ASCII values---
		 * special characters: 	32 to 47
		 * numbers: 		48 to 57
		 * special characters: 	58 to 64
		 * letters uppercase: 	65 to 90
		 * special characters: 	91 to 96
		 * letters lowercase: 	97 to 122
		 * special characters: 	123 to 126
		 * */
	} else if( argc == 4 ) {
		password = argv[1];
		first_char = atoi( argv[2] );
		last_char = atoi( argv[3] );
	} else {
		printf("usage: breaker <password> <first_ch> <last_ch>\n");
		printf("default: breaker A$4c 32 126\n");
		printf("exemple to break the binary password 1101000:\n");
		printf( "breaker 1101000 48 49\n" );
		exit( 0 );
	}
	
	int length = strlen(password);	
	int sz_in_bytes = sizeof(int);

	int *h_crypted = (int *)malloc(sizeof(int));

	int crypted_to_break= get_keypair(password, length, first_char, last_char);
	
	h_crypted = &crypted_to_break;

	int *d_crypted=(int *)malloc(sizeof(int));
	
	printf( "*running parameters*\n" );
	printf( " -password length:\t%lu digits\n", strlen(password) );
	printf( " -password length:\t%s digits\n", password);
	printf( " -digits:\t\tfrom -%c- to -%c-\n", first_char, last_char );
	printf(	" -crypted to break:\t%d\n", crypted_to_break);
	
	t1 = clock();
	
	hipMalloc((void**)&d_crypted, sz_in_bytes);
 
        hipMemcpy(d_crypted, h_crypted, sz_in_bytes, hipMemcpyHostToDevice);

	dim3 nBlocks;                                                                                
        dim3 nThperBlock;

	nBlocks.x = 16;
	nThperBlock.x = 1024;

	int loop_size = last_char - first_char;
	double max_iter = powl(loop_size, length);
		
	kernel<<< nBlocks , nThperBlock >>>(d_crypted, length, first_char, last_char, max_iter);
	hipDeviceSynchronize();
	
	t2 = clock();

	hipMemcpy(h_crypted, d_crypted, sz_in_bytes, hipMemcpyDeviceToHost);   
        hipFree(d_crypted);

	
	float period = (t2-t1)/CLOCKS_PER_SEC;
	if( period < 60 ){
		printf( "time: %.1fs \n", period );
	}else{
		printf( "time: %.1fmin \n", period/60 );
	}
	
	return EXIT_SUCCESS;
}

