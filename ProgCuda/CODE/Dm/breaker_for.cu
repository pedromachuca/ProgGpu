#include "hip/hip_runtime.h"
//nvcc -ccbin clang-3.8 Ex3.cu -o Ex3



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
//#define __USE_GNU
#include <crypt.h>

#define SIZE 102400

__global__ void kernel(char *crypted, int length, int first_char, int last_char, int N){

	//int loop_size = last_char - first_char;
	//int cryptlen = strlen(crypted);
	//int max_iter = powl(loop_size, length);
	char tab[4];
	//char tab[length];
	tab[length]='\0';
	int j;

	for(j=0; j<length; j++) tab[j] = first_char;

	long double i;
	//int ret = -1;
	printf("max_iter = %lu \n", (unsigned long) max_iter);	

	for(i=0; i<max_iter; i++)
	{
		if( !strcmp( crypted, crypt( tab, "salt" ) ) ) {
			printf( "password found: %s\n", tab );
			//return i;
		}	
		tab[0]++;
		for(j=0; j<length-1; j++)
		{
			if(last_char == tab[j])
			{
				tab[j] = first_char;
				tab[j+1]++;
			}
		}		
	}	
	//return i;
}
int init(int length, int first_char, int last_char){
	
	int loop_size = last_char - first_char;
	int cryptlen = strlen(crypted);
	int max_iter = powl(loop_size, length);
	char tab[length];
	
}
/*
int search_all_1( char* crypted, int length, int first_char, int last_char ){
	int loop_size = last_char - first_char;
	int cryptlen = strlen(crypted);
	int max_iter = powl(loop_size, length);
	char tab[length];
	tab[length]='\0';
	int j;
	for(j=0; j<length; j++) tab[j] = first_char;

	long double i;
	int ret = -1;
	printf("max_iter = %lu \n", (unsigned long) max_iter);	

	for(i=0; i<max_iter; i++)
	{
		if( !strcmp( crypted, crypt( tab, "salt" ) ) ) {
			printf( "password found: %s\n", tab );
			return i;
		}	
		tab[0]++;
		for(j=0; j<length-1; j++)
		{
			if(last_char == tab[j])
			{
				tab[j] = first_char;
				tab[j+1]++;
			}
		}		
	}	
	return i;
}*/


int main( int argc, char** argv ) {
	
	char* password; 
	struct timeval t1;
	struct timeval t2; 
	int first_char, last_char;
	//int cmp;
	
	if( argc == 1 ) {
		password = "A$4c";
		first_char = 32;
		last_char = 126;
		/* ---ASCII values---
		 * special characters: 	32 to 47
		 * numbers: 		48 to 57
		 * special characters: 	58 to 64
		 * letters uppercase: 	65 to 90
		 * special characters: 	91 to 96
		 * letters lowercase: 	97 to 122
		 * special characters: 	123 to 126
		 * */
	} else if( argc == 4 ) {
		password = argv[1];
		first_char = atoi( argv[2] );
		last_char = atoi( argv[3] );
	} else {
		printf("usage: breaker <password> <first_ch> <last_ch>\n");
		printf("default: breaker A$4c 32 126\n");
		printf("exemple to break the binary password 1101000:\n");
		printf( "breaker 1101000 48 49\n" );
		exit( 0 );
	}
	
	int length = strlen(password);
	
	char* crypted0 = crypt( password, "salt" );
	
	char* h_crypted = (char*) malloc( (strlen(crypted0)+1)*sizeof(char) );
	char* d_crypted = (char*) malloc( (strlen(crypted0)+1)*sizeof(char) );
	
	strcpy( h_crypted, crypted0 );

	printf( "*running parameters*\n" );
	printf( " -password length:\t%d digits\n", strlen(password) );
	printf( " -digits:\t\tfrom -%c- to -%c-\n", first_char, last_char );
	printf(	" -crypted to break:\t%s\n", h_crypted );
	
	int sz_in_byte =  strlen(h_crypted)*sizeof(char);

	hipMalloc((void**)&d_crypted, sz_in_byte);
	hipMemcpy(d_crypted, h_crypted, sz_in_byte, hipMemcpyHostToDevice);
	
	dim3 nBlocks;                                                                                
	dim3 nThperBlock;
	
	nBlocks.x = 16;
	nThperBlock.x = 1024;

	gettimeofday(&t1, NULL);
		
	kernel<<< nBlocks , nThperBlock >>>(d_crypted, length, first_char, last_char, SIZE);
	
	hipDeviceSynchronize();
	hipMemcpy(h_crypted, d_crypted, sz_in_byte, hipMemcpyDeviceToHost);
	hipFree(d_crypted);

	//cmp = ??
	//cmp = search_all_1( crypted, strlen( password ), first_char, last_char );
	gettimeofday(&t2, NULL);

//	double period =(double)((int)(t2.tv_sec-t1.tv_sec))+((double)(t2.tv_usec-t1.tv_usec))/1000000;  

	printf( "time: %dmin %.3fs \n", (int)((t2.tv_sec-t1.tv_sec))/60, (double)((int)(t2.tv_sec-t1.tv_sec)%60)+((double)(t2.tv_usec-t1.tv_usec))/1000000 );
	//printf( "#tries: %d\n", cmp );
	//printf( "=> efficiency: %.f tries/s\n", (double)cmp/period );

	return EXIT_SUCCESS;
}
