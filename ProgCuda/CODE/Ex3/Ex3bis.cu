#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SIZE 102400
#define MOD 102399
#define STEP 1

/* ARRAY A INITIALIZER */
void init_a(int * a)
{
    int i;
    for(i=0; i<SIZE; i++)
    {
        a[i] = 1;
    }
}

/* ARRAY B INITIALIZER */
void init_b(int * b)
{
	int i, j;

	j=0;

	for(i=0; i<SIZE-1; i++)
	{
		b[j] = i;
		j = (j+STEP)%MOD;
	}	

    b[SIZE-1] = SIZE-1;
}

/* CHECKING A VALUES */
int check_a(int * a)
{
    int i;
    int correct = 1;
	for(i=0; i<SIZE; i++)
	{
		if(a[i] != (i+1)) 
		{
         
			correct = 0;
		} 
	}	

    return correct;
}


/* CUDA FUNCTION */
__global__ void mykernel(int * a, int * b, int N)
{
	/*
	int i =blockIdx.x * blockDim.x + threadIdx.x;
    int total = (blockDim.x * gridDim.x);
    for(int j = i; j < N; j += total){
        a[b[j]] += b[j];
    }*/
	//Method prof :
	int index = threadIdx.x;
	int tmp;
	for(;index <N; index+=blockDim.x){
		tmp = b[index];
		a[tmp] = a[tmp]+tmp;
	}
}


int main(int argc, char * argv[])
{
	int sz_in_bytes = SIZE*sizeof(int);

	int * h_a = (int *)malloc(sz_in_bytes);
	int * h_b = (int *)malloc(sz_in_bytes);
	
	int *d_a, *d_b;

    init_a(h_a);
	init_b(h_b);
	
	hipMalloc((void**)&d_a, sz_in_bytes);
	hipMalloc((void**)&d_b, sz_in_bytes);

	hipMemcpy(d_a, h_a, sz_in_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sz_in_bytes, hipMemcpyHostToDevice);

	dim3 nBlocks;
	dim3 nThperBlock;

	nBlocks.x = 1;
	
	nThperBlock.x = 1024;

	struct timeval tv_start, tv_stop;
	gettimeofday(&tv_start, NULL);

	mykernel<<< nBlocks , nThperBlock >>>(d_a, d_b, SIZE);

	hipDeviceSynchronize();

	gettimeofday(&tv_stop, NULL);

	hipMemcpy(h_a, d_a, sz_in_bytes, hipMemcpyDeviceToHost);
	
	hipFree(d_a);
	hipFree(d_b);	

	int correct = check_a(h_a);;
	
	if(0 == correct)
	{
		printf("\n\n ******************** \n ***/!\\ ERROR /!\\ *** \n ******************** \n\n");
	}
	else
	{
		printf("\n\n ******************** \n ***** SUCCESS! ***** \n ******************** \n\n");
	}
	free(h_a);
	free(h_b);

	int nsec = tv_stop.tv_sec - tv_start.tv_sec;
	int nusec = tv_stop.tv_usec - tv_start.tv_usec;
	if(nusec <0){
		nusec = nusec + 1000000;
		nsec = nsec -1;
	}
	printf("time = %d s,%d us", nsec, nusec);
	return 1;
}
