
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define checkCudaErrors(val)\
	fprintf(stderr, "CUDA error at %s:%d (%s) \n", __FILE__, __LINE__, hipGetErrorString(val));

//Par rapport a la question 7 N = 1000 et nb thread = 640 
//		=>si on fait 2 x nb_thread alors 1280 threads > N peut causer bufferoverflow/seg fault

__global__ void kernel(double *a, double *b, double *c, int N)
{
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
	//Q 8 :
	/*
    int i = 2*(blockIdx.x * blockDim.x + threadIdx.x);
	if(i<N-1){
		c[i] = a[i] + b[i];
		c[i+1] = a[i+1] + b[i+1];
	}*/
	//Q 8 second way :
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int totalthreads = (blockDim.x * gridDim.x);
	c[i] = a[i] + b[i];
	if(i<(N-totalthreads)){
		c[i+totalthreads] = a[i+totalthreads] + b[i+totalthreads];
	}


}

int main(int argc, char **argv)
{
    int N = 1000;
    int sz_in_bytes = N*sizeof(double);

    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;

    h_a = (double*)malloc(sz_in_bytes);
    h_b = (double*)malloc(sz_in_bytes);
    h_c = (double*)malloc(sz_in_bytes);

    // Initiate values on h_a and h_b
    for(int i = 0 ; i < N ; i++)
    {
	h_a[i] = 1./(1.+i);
	h_b[i] = (i-1.)/(i+1.);
    }

    checkCudaErrors(hipMalloc((void**)&d_a, sz_in_bytes));
    checkCudaErrors(hipMalloc((void**)&d_b, sz_in_bytes));
    checkCudaErrors(hipMalloc((void**)&d_c, sz_in_bytes));

    checkCudaErrors(hipMemcpy(d_a, h_a, sz_in_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, sz_in_bytes, hipMemcpyHostToDevice));

    dim3  dimBlock(64, 1, 1);
    dim3  dimGrid(10, 1, 1);
    kernel<<<dimGrid , dimBlock>>>(d_a, d_b, d_c, N);

    checkCudaErrors(hipMemcpy(h_c, d_c, sz_in_bytes, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));

    // Verifying
    double err = 0, norm = 0;
    for(int i = 0 ; i < N ; i++)
    {
		double err_loc = fabs(h_c[i] - (h_a[i]+h_b[i]));
		err  += err_loc;
		norm += fabs(h_c[i]);
    }
    if (err/norm < 1.e-16)
    {
		printf("SUCCESS (Relative error : %.3e)\n", err/norm);
    }
    else
    {
		printf("ERROR (Relative error : %.3e)\n", err/norm);
    }

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

